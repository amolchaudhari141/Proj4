/*
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/*
 To compile: 
    /usr/local/cuda/bin/nvcc -arch=sm_30 reduction_kernel.cu
 To run with the array size 2^20, expo dist mean 5, and init seed 17:
    ./a.out 20 5 17
 */
#include <stdio.h>
#include <stdlib.h>



extern "C" void reduce_wrapper(int *n, int *mean, int *seed, double *max_val);



template<class T>
struct SharedMemory
{
    __device__ inline operator       T *()
    {
        extern __shared__ int __smem[];
        return (T *)__smem;
    }

    __device__ inline operator const T *() const
    {
        extern __shared__ int __smem[];
        return (T *)__smem;
    }
};

//run CPU version

double 
reduceCPU(double *data, int size)
{   double cpu_max = -1.0;
    for (unsigned int i = 1; i < size; i++)
    {  
        if (data[i] > cpu_max)
            cpu_max = data[i];           
    }
    return cpu_max;
}

/*
    Parallel sum reduction using shared memory
    - takes log(n) steps for n input elements
    - uses n threads
    - only works for power-of-2 arrays
*/

/*
    This version uses sequential addressing -- no divergence or bank conflicts.
*/
__global__ void
reduce(double *g_idata, double *max, int *mutex, unsigned int n)
{
    double *sdata = SharedMemory<double>();

    // load shared mem
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int stride = gridDim.x*blockDim.x;
    unsigned int offset = 0;
    // sdata[tid] = (i < n) ? g_idata[i] : 0;
    float local_max = -1.0;
    while( i+offset < n){
        local_max = fmaxf(local_max, g_idata[i+offset]);
        offset += stride;
    }
    sdata[tid] = local_max;
    __syncthreads();

    // do reduction in shared mem
    for (unsigned int s=blockDim.x/2; s>0; s>>=1)
    {
        if (tid < s)
        {
            sdata[tid] = fmaxf(sdata[tid], sdata[tid + s]);
        }
        __syncthreads();
    }

    // write result for this block to global mem
    if (tid == 0){
    while(atomicCAS(mutex, 0, 1) != 0); //lock
    *max = fmaxf(*max, sdata[0]);
    atomicExch(mutex, 0); //unlock
    }
}


// CUDA Runtime
#include <hip/hip_runtime.h>

#define checkCudaErrors(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

void
reduce_wrapper(int *n, int *mean, int *seed, double *max_val)
{
        // int n = atoi(argv[1]); 
        // int mean = atoi(argv[2]);
        // int seed = atoi(argv[3]); 
        int size = 1 << *n;    // number of elements to reduce
        int maxThreads = 256;  // number of threads per block

        // create random input data on CPU
        unsigned int bytes = size * sizeof(double);

        double *h_idata = (double *) malloc(bytes);
        double *h_max = (double *) malloc(sizeof(double));  /*value to hold result from device */

        srand48(*seed);
        for (int i=0; i<size; i++)
        {
                // h_idata[i] = 1.0; // for testing
                // expo dist with mean 5.0
                h_idata[i] = -(*mean) * log(drand48());
        }

        int numBlocks = size / maxThreads;
        int numThreads = size;

        int smemSize = maxThreads * sizeof(double);

        // allocate device memory and data
        double  *d_idata = NULL;
        double *d_max;
        int *d_mutex;   /*mutex write protection in shared memory */
     

        checkCudaErrors(hipMalloc((void **) &d_idata, bytes));
        checkCudaErrors(hipMalloc((void **) &d_max, sizeof(double)));
        checkCudaErrors(hipMalloc((void **) &d_mutex, sizeof(int)));

        // initialize d_max to be 0
        checkCudaErrors(hipMemset(d_max, 0, sizeof(double)));

        //set Mutex to be unlocked
        checkCudaErrors(hipMemset(d_mutex, 0, sizeof(int))); 

        // copy data directly to device memory
        checkCudaErrors(hipMemcpy(d_idata, h_idata, bytes, hipMemcpyHostToDevice));
        //call reduce
        reduce<<<numBlocks,maxThreads,smemSize>>>(d_idata, d_max, d_mutex, numThreads);

        checkCudaErrors(hipMemcpy(h_max, d_max, sizeof(double), hipMemcpyDeviceToHost));

        printf("GPU max : %f\n", *h_max);

        //return h_max to call function in C
        *max_val = *h_max;

        //free cuda memory
        checkCudaErrors(hipFree(d_idata));
        checkCudaErrors(hipFree(d_max));
        
        double cpu_max;
        cpu_max = reduceCPU(h_idata, size);
   
        printf("CPU max : %f\n", cpu_max);

}
